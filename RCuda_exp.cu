#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <Rinternals.h>    
#include <R.h>
#include <math.h>

// treat it as C code
extern "C" {
  SEXP vexp(SEXP x);
}

__global__ void dexp( double *a, double *b, int n )
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // handle the data at this index
  while (tid < n) {
    b[tid] = exp(a[tid]);
    tid += blockDim.x * gridDim.x;
  }
  //printf("Value of *ip variable: %f\n", a[tid] );

}


SEXP vexp(SEXP x) {
  SEXP res;
  int nx;
  nx = length(x);

  PROTECT(res = allocVector(REALSXP,nx));
  
  // Turn vectors into C objects
  double *h_x = REAL(x);
  double *h_res = REAL(res);
  
  // Create pointers for device
  double *d_x, *d_res; // Pointer for the device (GPU)
    
  // Allocate memory on GPU
  int bytes = nx * sizeof(double);
  hipMalloc(&d_x, bytes );
  hipMalloc(&d_res, bytes );

  // Copy vectors x, y and res to GPU
  hipMemcpy( d_x, h_x, bytes, hipMemcpyHostToDevice );
  
  // Set number of operations
  // Run code
  dexp<<<256,256>>>(d_x, d_res, nx);

  // Load result back from GPU
  hipMemcpy( h_res, d_res, bytes, hipMemcpyDeviceToHost );

  // Free memory on the GPU
  hipFree( d_x );
  hipFree( d_res );

  // Unprotect res. Allows R to clean it up.
  UNPROTECT(1);
  
  return res;
}
