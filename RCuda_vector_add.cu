#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <Rinternals.h>    
#include <R.h>

// treat it as C code
extern "C" {
  SEXP vadd(SEXP x, SEXP y);
}

__global__ void add( double *a, double *b, double *c, int n )
{
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // handle the data at this index
  while (tid < n) {
    c[tid] = a[tid] + b[tid];
    tid += blockDim.x * gridDim.x;
  }
  //printf("Value of *ip variable: %f\n", a[tid] );

}


SEXP vadd(SEXP x, SEXP y) {
  SEXP res;
  int nx;
  nx = length(x);

  PROTECT(res = allocVector(REALSXP,nx));
  
  // Turn vectors into C objects
  double *h_x = REAL(x);
  double *h_y = REAL(y);
  double *h_res = REAL(res);

  
  //printf("Value of x_0 variable: %f\n", h_x[0] );
  
  // Create pointers for device
  double *d_x, *d_y, *d_res; // Pointer for the device (GPU)
    
  // Allocate memory on GPU
  int bytes = nx * sizeof(double);
  hipMalloc(&d_x, bytes );
  hipMalloc(&d_y, bytes );
  hipMalloc(&d_res, bytes );

  // Copy vectors x, y and res to GPU
  hipMemcpy( d_x, h_x, bytes, hipMemcpyHostToDevice );
  hipMemcpy( d_y, h_y, bytes, hipMemcpyHostToDevice );
  
  // Set number of operations
  // Run code
  add<<<128,128>>>(d_x, d_y, d_res, nx);
  //add<<<nx,1>>>(d_x, d_y, d_res, nx);

  // Load result back from GPU
  hipMemcpy( h_res, d_res, bytes, hipMemcpyDeviceToHost );

  // Free memory on the GPU
  hipFree( d_x );
  hipFree( d_y );
  hipFree( d_res );

  // Unprotect res. Allows R to clean it up.
  UNPROTECT(1);
  
  return res;
}
